#include "hip/hip_runtime.h"
#include <chrono> 
#include <iostream>
#include <string>
#include <thread>
#include <vector>

#include "util_cuda.h"

void run_newton(unsigned n_epochs,
                unsigned n_streams,
                unsigned n_kernels_per_stream,
                unsigned array_size,
                unsigned block_dim,
                double* x,
                bool multithreaded)
{
    // n_kernels: total number of kernel launch we will do over all the streams
    const unsigned n_kernels = n_kernels_per_stream * n_streams;
    // array_size / kernels : size of the portion of the array that each kernel launch should be processing
    const unsigned k_arr_size = array_size/n_kernels;
    // rounded up division to know how many gpu thread blocks to spawn per kernel launch
    const unsigned grid_dim = (k_arr_size-1)/block_dim + 1;
    // leftover size for the array
    const unsigned k_arr_size_last = array_size - (k_arr_size * (n_kernels-1));
    // size of the last grid of last kernel launch
    const unsigned grid_dim_last = (k_arr_size_last-1)/block_dim + 1;

    hipStream_t* streams = new hipStream_t[n_streams];
    for (int i = 0; i < n_streams; i++) {
        hipStreamCreate(&streams[i]);
    }

    //
    auto thread_runner = [&](unsigned stream_idx) {
        for (unsigned k = 0; k < n_kernels_per_stream; ++k) {
            unsigned kernel_idx   = k + stream_idx * n_kernels_per_stream;
            unsigned kernel_start = kernel_idx * k_arr_size;

            auto launch_arr_size =  (kernel_idx == (n_kernels-1)) ? k_arr_size_last : k_arr_size;
            auto launch_grid_dim =  (kernel_idx == (n_kernels-1)) ? grid_dim_last : grid_dim;

            kernels::newton<<<launch_grid_dim, block_dim, 0, streams[stream_idx]>>>(x+kernel_start, launch_arr_size);
        }

    };

    if (multithreaded) {
        for (unsigned i = 0; i < n_epochs; ++i) {
            std::vector<std::thread> threads;
            for (unsigned stream_idx = 0; stream_idx < n_streams; ++stream_idx) {
                threads.push_back(std::thread(thread_runner, stream_idx));
            }

            // wait for all the cpu threads to have finished
            for (unsigned stream_idx = 0; stream_idx < n_streams; ++stream_idx) {
                threads[stream_idx].join();
            }

            // wait for all gpu kernels to have completed
            device_synch();
        }
    }
    else {
        for (unsigned i = 0; i < n_epochs; ++i) {
            for (unsigned stream_idx = 0; stream_idx < n_streams; ++stream_idx) {
                thread_runner(stream_idx);
            }
            // wait for all gpu kernels to have completed
            device_synch();
        }
    }

    for (int i = 0; i < n_streams; i++) {
        hipStreamDestroy(streams[i]);
    }
    delete[] streams;
}

int main(int argc, char** argv) {
    const unsigned n_epochs               = read_arg(argc, argv, 1, 10);
    const unsigned n_streams              = read_arg(argc, argv, 2, 1);
    const unsigned n_kernels_per_stream   = read_arg(argc, argv, 3, 4);
    const unsigned pow                    = read_arg(argc, argv, 4, 20);
    const unsigned block_dim              = read_arg(argc, argv, 5, 128);
    const unsigned multithreaded          = read_arg(argc, argv, 6, 0);

    const unsigned array_size = 2 << pow;

    std::cout << "array_size          = " << array_size  << std::endl;
    std::cout << "epochs              = " << n_epochs << std::endl;
    std::cout << "streams             = " << n_streams << std::endl;
    std::cout << "kernels_per_stream  = " << n_kernels_per_stream << std::endl;
    std::cout << "block_dim           = " << block_dim << std::endl;
    std::cout << "threading           = " << (multithreaded ? std::to_string(n_streams)+" threads" : "1 thread") << std::endl;

    // Run the newton kernel a bunch of times on a larger array to "warm up"
    {
        unsigned ni = 2<<24;
        unsigned grid_dim = (ni-1)/block_dim + 1;
        double* xhi = malloc_host<double>(ni);
        double* xdi = malloc_device<double>(ni);
        std::fill(xhi, xhi+ni, 2.3);
        copy_to_device<double>(xhi, xdi, ni);
        for (auto i=0; i<100; ++i) {
            kernels::newton<<<grid_dim, block_dim>>>(xdi, ni);
        }
        std::free(xhi);
        free_device(xdi);
    }

    double* xh = malloc_host<double>(array_size);
    double* yh = malloc_host<double>(array_size);

    double* xd = malloc_device<double>(array_size);
    double* yd = malloc_device<double>(array_size);

    std::fill(xh, xh+array_size, 2.0);
    std::fill(yh, yh+array_size, 1.0);

    copy_to_device<double>(xh, xd, array_size);
    copy_to_device<double>(yh, yd, array_size);

    device_synch();
    auto start = std::chrono::system_clock::now();

    start_gpu_prof();

    run_newton(n_epochs, n_streams, n_kernels_per_stream, array_size, block_dim, xd, multithreaded);

    stop_gpu_prof();

    auto end = std::chrono::system_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    std::cout << "Runtime    = " << elapsed.count() << "us \n";
    std::cout << "Throughput = " << n_epochs * (array_size * sizeof(double)) / (double)(elapsed.count()) << "MB/s \n";

    std::free(xh);
    std::free(yh);
    free_device(xd);
    free_device(yd);

    return 0;
}

